#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include ""
#include "device_helpers.hpp"

namespace emida
{

/*  Optimization idea:
	The amount of work the threads have to do looks like this:
	64	72	80	88	96	88	80	72	64
	72	81	90	99	108	99	90	81	72
	80	90	100	110	120	110	100	90	80
	88	99	110	121	132	121	110	99	88
	96	108	120	132	144	132	120	108	96
	88	99	110	121	132	121	110	99	88
	80	90	100	110	120	110	100	90	80
	72	81	90	99	108	99	90	81	72
	64	72	80	88	96	88	80	72	64
	Threads near 0 offset do the most work.
	So some threads in the same warp/block may do much more work than others.
	Assign pixels to threads in a way that threads from the same thread do
	the same amount of work?
*/

template<typename T, typename RES>
__global__ void cross_corr(
	const T* __restrict__ pics,
	const T* __restrict__ ref,
	RES* __restrict__ res,
	size2_t size,
	size2_t res_size,
	size_t ref_slices,
	size_t batch_size)
{
	size_t whole_x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t cuda_y = blockIdx.y * blockDim.y + threadIdx.y;
	
	//number of picture that this thread computes
	size_t slice_num = whole_x / res_size.x;

	if (slice_num >= ref_slices || cuda_y >= res_size.y)
		return;

	size2_t slice_pos = { whole_x % res_size.x, cuda_y };
	size_t ref_num = slice_num % ref_slices;
	

	size2_t r = (res_size - 1) / 2;

	vec2<int> shift = { (int)slice_pos.x - (int)r.x, (int)slice_pos.y - (int)r.y };
	
	ref += ref_num * size.area();
	pics += slice_num * size.area();
	res += slice_num * res_size.area();

	
	for (size_t i = 0; i < batch_size; ++i)
	{
		size_t x_end = shift.x < 0 ? size.x : size.x - shift.x;
		size_t y_end = shift.y < 0 ? size.y : size.y - shift.y;

		//control flow divergency in following fors??
		RES sum = 0;
		for (size_t y = shift.y >= 0 ? 0 : -shift.y; y < y_end; ++y)
		{
			for (size_t x = shift.x >= 0 ? 0 : -shift.x; x < x_end; ++x)
			{
				int x_shifted = x + shift.x;
				int y_shifted = y + shift.y;

				sum += pics[y_shifted * size.x + x_shifted] * ref[y * size.x + x];
			}
		}


		res[slice_pos.pos(res_size.x)] = sum;

		pics += ref_slices * size.area();
		res += ref_slices * res_size.area();
	}
}

template<typename T, typename RES>
void run_cross_corr(const T* pic_a, const T* pic_b, RES* res, vec2<size_t> size, vec2<size_t> res_size, size_t ref_slices, size_t batch_size)
{	
	dim3 block_size(16, 16);
	dim3 grid_size(div_up(res_size.x * ref_slices, block_size.x), div_up(res_size.y, block_size.y));
	cross_corr<T, RES> <<<grid_size, block_size>>> (pic_a, pic_b, res, size, res_size, ref_slices, batch_size);
}


template void run_cross_corr<int, int>(
	const int*,
	const int*,
	int* res,
	vec2<size_t> size,
	vec2<size_t> res_size,
	size_t,
	size_t);

template void run_cross_corr<double, double>(
	const double*,
	const double*,
	double* res,
	vec2<size_t> size,
	vec2<size_t> res_size,
	size_t,
	size_t);

template void run_cross_corr<float, float>(
	const float*,
	const float*,
	float* res,
	vec2<size_t> size,
	vec2<size_t> res_size,
	size_t,
	size_t);

template<typename T>
__device__ __inline__ void copy_subregion(const T * __restrict__ src, size_t src_stride, T* __restrict__ dest, size2_t dest_size, size2_t region_pos)
{
	for (size_t x = threadIdx.x; x < dest_size.x; x += blockDim.x)
		for (size_t y = threadIdx.y; y < dest_size.y; y += blockDim.y)
		{
			dest[y * dest_size.x + x] = src[(y + region_pos.y) * src_stride + (x + region_pos.x)];
		}
}

template<typename T, typename RES>
__global__ void cross_corr_opt(
	const T* __restrict__ pics,
	const T* __restrict__ ref,
	RES* __restrict__ res,
	size2_t size,
	size2_t res_size,
	size_t ref_slices,
	size_t batch_size)
{
	size_t whole_x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t cuda_y = blockIdx.y * blockDim.y + threadIdx.y;

	size_t block_grid_width = div_up(size.x, gridDim.x);
	size2_t pic_block_pos = size2_t::from_id(blockIdx.x, block_grid_width) * blockDim;
	size2_t ref_block_pos = size2_t::from_id(blockIdx.y, block_grid_width) * blockDim;

	size2_t reg_size = { blockDim.x / 2, blockDim.y / 2 };
	size2_t res_reg_size = { blockDim.x - 1, blockDim.y - 1 };


	T* smem = shared_memory_proxy<T>();
	T* pic_reg = smem;
	T* ref_reg = smem + reg_size.area();
	T* res_reg = smem + 2 * + reg_size.area();

	copy_subregion(pics, size.x, pic_reg, size2_t{ blockDim.x, blockDim.y }, pic_block_pos);
	copy_subregion(ref, size.x, pic_reg, size2_t{ blockDim.x, blockDim.y }, pic_block_pos);

	__syncthreads();

	size2_t r = (res_size - 1) / 2;
	size2_t reg_r = reg_size - 1;

	vec2<int> shift = { (int)threadIdx.x - (int)reg_r.x, (int)threadIdx.y - (int)reg_r.y };

	//ref += ref_num * size.area();
	//pics += slice_num * size.area();
	//res += slice_num * res_size.area();


	size_t x_end = shift.x < 0 ? blockDim.x : blockDim.x - shift.x;
	size_t y_end = shift.y < 0 ? blockDim.y : blockDim.y - shift.y;

	//control flow divergency in following fors??
	RES sum = 0;
	for (size_t y = shift.y >= 0 ? 0 : -shift.y; y < y_end; ++y)
	{
		for (size_t x = shift.x >= 0 ? 0 : -shift.x; x < x_end; ++x)
		{
			int x_shifted = x + shift.x;
			int y_shifted = y + shift.y;

			sum += pic_reg[y_shifted * reg_size.x + x_shifted] * ref_reg[y * reg_size.x + x];
		}
	}

	
	RES* res_ptr = res + (pic_block_pos - ref_block_pos + size2_t{ threadIdx.x, threadIdx.y }).pos(res_size.x);
	atomicAdd(res_ptr, sum);

	//pics += ref_slices * size.area();
	//res += ref_slices * res_size.area();

}

template<typename T, typename RES>
void run_cross_corr_opt(
	const T* pic_a,
	const T* pic_b,
	RES* res,
	size2_t size,
	size2_t res_size,
	size2_t block_size,
	size_t ref_slices,
	size_t batch_size)
{
	//smem: 6 * blockDim.x * blockDim.y
	dim3 block_dim(block_size.x, block_size.y);
	size_t blocks = div_up(size.x, block_size.x) * div_up(size.y, block_size.y);
	dim3 grid_size(blocks, blocks);
	cross_corr_opt<T, RES> <<<grid_size, block_dim, 2 * block_size.area() >>> (pic_a, pic_b, res, size, res_size, ref_slices, batch_size);
}

template void run_cross_corr_opt<double, double>(
	const double*,
	const double*,
	double* res,
	size2_t size,
	size2_t res_size,
	size2_t block_size,
	size_t,
	size_t);


}