#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.cuh"
#include ""
#include "device_helpers.hpp"

namespace emida
{

/*  Optimization idea:
	The amount of work the threads have to do looks like this:
	64	72	80	88	96	88	80	72	64
	72	81	90	99	108	99	90	81	72
	80	90	100	110	120	110	100	90	80
	88	99	110	121	132	121	110	99	88
	96	108	120	132	144	132	120	108	96
	88	99	110	121	132	121	110	99	88
	80	90	100	110	120	110	100	90	80
	72	81	90	99	108	99	90	81	72
	64	72	80	88	96	88	80	72	64
	Threads near 0 offset do the most work.
	So some threads in the same warp/block may do much more work than others.
	Assign pixels to threads in a way that threads from the same thread do
	the same amount of work?
*/

template<typename T, typename RES>
__global__ void cross_corr(
	const T* __restrict__ pics,
	const T* __restrict__ ref,
	RES* __restrict__ res,
	size2_t size,
	size2_t res_size,
	size_t ref_slices,
	size_t batch_size)
{
	size_t whole_x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t cuda_y = blockIdx.y * blockDim.y + threadIdx.y;
	
	//number of picture that this thread computes
	size_t slice_num = whole_x / res_size.x;

	if (slice_num >= ref_slices || cuda_y >= res_size.y)
		return;

	size2_t slice_pos = { whole_x % res_size.x, cuda_y };
	size_t ref_num = slice_num % ref_slices;
	

	size2_t r = (res_size - 1) / 2;

	vec2<int> shift = { (int)slice_pos.x - (int)r.x, (int)slice_pos.y - (int)r.y };
	
	ref += ref_num * size.area();
	pics += slice_num * size.area();
	res += slice_num * res_size.area();

	
	for (size_t i = 0; i < batch_size; ++i)
	{
		size_t x_end = shift.x < 0 ? size.x : size.x - shift.x;
		size_t y_end = shift.y < 0 ? size.y : size.y - shift.y;

		//control flow divergency in following fors??
		RES sum = 0;
		for (size_t y = shift.y >= 0 ? 0 : -shift.y; y < y_end; ++y)
		{
			for (size_t x = shift.x >= 0 ? 0 : -shift.x; x < x_end; ++x)
			{
				int x_shifted = x + shift.x;
				int y_shifted = y + shift.y;

				sum += pics[y_shifted * size.x + x_shifted] * ref[y * size.x + x];
			}
		}


		res[slice_pos.pos(res_size.x)] = sum;

		pics += ref_slices * size.area();
		res += ref_slices * res_size.area();
	}
}

template<typename T, typename RES>
void run_cross_corr(const T* pic_a, const T* pic_b, RES* res, vec2<size_t> size, vec2<size_t> res_size, size_t ref_slices, size_t batch_size)
{	
	dim3 block_size(16, 16);
	dim3 grid_size(div_up(res_size.x * ref_slices, block_size.x), div_up(res_size.y, block_size.y));
	cross_corr<T, RES> <<<grid_size, block_size>>> (pic_a, pic_b, res, size, res_size, ref_slices, batch_size);
}


template void run_cross_corr<int, int>(
	const int*,
	const int*,
	int* res,
	vec2<size_t> size,
	vec2<size_t> res_size,
	size_t,
	size_t);

template void run_cross_corr<double, double>(
	const double*,
	const double*,
	double* res,
	vec2<size_t> size,
	vec2<size_t> res_size,
	size_t,
	size_t);

template void run_cross_corr<float, float>(
	const float*,
	const float*,
	float* res,
	vec2<size_t> size,
	vec2<size_t> res_size,
	size_t,
	size_t);

template<typename T>
__device__ __inline__ void copy_subregion(const T * __restrict__ src, size2_t src_size, T* __restrict__ dest, size2_t dest_size, size2_t region_pos)
{
	for (size_t y = threadIdx.y; y < dest_size.y; y += blockDim.y)
		for (size_t x = threadIdx.x; x < dest_size.x; x += blockDim.x)
		{
			dest[y * dest_size.x + x] = x + region_pos.x < src_size.x && y + region_pos.y < src_size.y
				? src[(y + region_pos.y) * src_size.x + (x + region_pos.x)]
				: 0;
		}
}

template<typename T, typename RES>
__global__ void cross_corr_opt(
	const T* __restrict__ pics,
	const T* __restrict__ ref,
	RES* __restrict__ res,
	size2_t size,
	size2_t res_size,
	size_t ref_slices,
	size_t batch_size)
{
	size2_t reg_size = { (blockDim.x + 1) / 2, (blockDim.y + 1) / 2 };
	size2_t res_reg_size = { blockDim.x - 1, blockDim.y - 1 };

	size_t block_grid_width = div_up(size.x, reg_size.x);
	
	size_t one_slice_blocks = gridDim.x / ref_slices;
	size_t slice_num = blockIdx.x / one_slice_blocks;
	size_t block_idx_x = blockIdx.x % one_slice_blocks;

	size2_t pic_block_pos = size2_t::from_id(block_idx_x, block_grid_width) * reg_size;
	size2_t ref_block_pos = size2_t::from_id(blockIdx.y, block_grid_width) * reg_size;
	/*if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		//printf("%d %d %d %d\n", blockDim.x, blockDim.y, gridDim.x, gridDim.y);

		printf("%d %d %d %d %d %d\n", blockIdx.x, (int)block_grid_width, (int)pic_block_pos.x, (int)pic_block_pos.y, (int)ref_block_pos.x, (int)ref_block_pos.y);
	}__syncthreads();*/
	
	T* smem = shared_memory_proxy<T>();
	T* pic_reg = smem;
	T* ref_reg = smem + reg_size.area();
	T* res_reg = smem + 2 * + reg_size.area();

	//ref += ref_num * size.area();
	ref += slice_num * size.area();
	pics += slice_num * size.area();
	res += slice_num * res_size.area();

	copy_subregion(pics, size, pic_reg, reg_size, pic_block_pos);
	copy_subregion(ref, size, ref_reg, reg_size, ref_block_pos);
	__syncthreads();
	/*if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 2 && blockIdx.y == 0)
	{
		printf("bb %d %d\n", (int)pic_block_pos.x, (int)pic_block_pos.y);
		for (size_t i = 0; i < reg_size.y; ++i)
		{
			for (size_t j = 0; j < reg_size.x; j++)
			{
				printf("%f ", pic_reg[i * reg_size.x + j]);
			}
			printf("\n");
		}
		for (size_t i = 0; i < reg_size.y; ++i)
		{
			for (size_t j = 0; j < reg_size.x; j++)
			{
				printf("%f ", ref_reg[i * reg_size.x + j]);
			}
			printf("\n");
		}
		
	}__syncthreads();*/

	//size2_t r = (res_size - 1) / 2;
	size2_t reg_r = reg_size - 1;
	vec2<int> res_r = { ((int)res_size.x - 1) / 2, ((int)res_size.y - 1) / 2 };
	vec2<int> shift = { (int)threadIdx.x - (int)reg_r.x, (int)threadIdx.y - (int)reg_r.y };
	vec2<int> block_shift = { (int)pic_block_pos.x - (int)ref_block_pos.x, (int)pic_block_pos.y - (int)ref_block_pos.y };
	vec2<int> res_pos = block_shift + shift + res_r;
	if (res_pos.x >= res_size.x || res_pos.y >= res_size.y)
		return;
	RES* res_ptr = res + (res_pos).pos(res_size.x);


	size_t x_end = shift.x < 0 ? reg_size.x : reg_size.x - shift.x;
	size_t y_end = shift.y < 0 ? reg_size.y : reg_size.y - shift.y;

	//control flow divergency in following fors??
	RES sum = 0;
	for (size_t y = shift.y >= 0 ? 0 : -shift.y; y < y_end; ++y)
	{
		for (size_t x = shift.x >= 0 ? 0 : -shift.x; x < x_end; ++x)
		{
			int x_shifted = x + shift.x;
			int y_shifted = y + shift.y;

			sum += pic_reg[y_shifted * reg_size.x + x_shifted] * ref_reg[y * reg_size.x + x];
		}
	}
	

	//vec2<int> block_shift = { (int)ref_block_pos.x - (int)pic_block_pos.x, (int)ref_block_pos.y - (int)pic_block_pos.y };

	//printf("%d %d %d %d %d %d %f %d %d %d %d %d %d\n", threadIdx.x, threadIdx.y, (int)pic_block_pos.x, (int)pic_block_pos.y, (int)ref_block_pos.x, (int)ref_block_pos.y, sum, (block_shift + shift + res_r).x,(block_shift + shift + res_r).y, shift.x, shift.y, res_r.x, res_r.y);
	
	atomicAdd(res_ptr, sum);

	//pics += ref_slices * size.area();
	//res += ref_slices * res_size.area();

}

template<typename T, typename RES>
void run_cross_corr_opt(
	const T* pics,
	const T* ref,
	RES* res,
	size2_t size,
	size2_t res_size,
	size2_t block_size,
	size_t ref_slices,
	size_t batch_size)
{
	dim3 block_dim(block_size.x, block_size.y);
	size2_t in_block_size = (block_size + 1) / 2;
	size_t blocks = div_up(size.x, in_block_size.x) * div_up(size.y, in_block_size.y);
	dim3 grid_size(blocks * ref_slices, blocks);
	cross_corr_opt<T, RES> <<<grid_size, block_dim, 2 * in_block_size.area() * sizeof(T) >>> (pics, ref, res, size, res_size, ref_slices, batch_size);
}

template void run_cross_corr_opt<double, double>(
	const double*,
	const double*,
	double* res,
	size2_t size,
	size2_t res_size,
	size2_t block_size,
	size_t,
	size_t);

template void run_cross_corr_opt<float, float>(
	const float*,
	const float*,
	float* res,
	size2_t size,
	size2_t res_size,
	size2_t block_size,
	size_t,
	size_t);


}