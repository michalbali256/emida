#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "device_helpers.hpp"
#include "kernels.cuh"

namespace emida
{
//        size
//      +-------+-------+-------+
//      +--+--+--+      +--+--+--+
//              +--+--+--+
//blockDim2  2 2  2 2  2  2  2  2  
//block: 1  2  3  4  5  6 7  8  9
template<typename T>
__global__ void maxarg_reduce(const T* data, data_index<T> * maxes, size_t size)
{
	data_index<T> * sdata = shared_memory_proxy<data_index<T>>();

	size_t tid = threadIdx.x;
	
	//number of blocks we need to process one picture
	size_t one_pic_blocks = div_up(size, blockDim.x);
	size_t pic_num = blockIdx.x / one_pic_blocks;
	size_t pic_block = blockIdx.x % one_pic_blocks;

	//if this is the last block that processes one picture(chunk)
	//and this thread would process sth out of the picture

	size_t i = pic_num * size + pic_block * blockDim.x + threadIdx.x;
	if (blockIdx.x % one_pic_blocks == one_pic_blocks - 1
		&& size % blockDim.x != 0
		&& threadIdx.x >= size % blockDim.x)
	{
		sdata[tid].data = 0;
		sdata[tid].index = i;
	}
	else
	{
		sdata[tid].data = data[i];
		sdata[tid].index = i;
	}
	
	__syncthreads();

	for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			if (sdata[tid + s].data > sdata[tid].data)
			{
				sdata[tid] = sdata[tid + s];
			}
		}
		__syncthreads();
	}
	
	if (tid == 0) maxes[blockIdx.x] = sdata[0];
}

template<typename T>
void run_maxarg_reduce(const T* data, data_index<T>* maxes, size_t size, size_t block_size, size_t batch_size)
{	
	size_t one_pic_blocks = div_up(size, block_size);
	size_t grid_size = one_pic_blocks * batch_size;
	maxarg_reduce<T> <<<grid_size, block_size, block_size * sizeof(data_index<T>)>>> (data, maxes, size);
}

template void run_maxarg_reduce<double>(const double* data, data_index<double>* maxes, size_t size, size_t block_size, size_t batch_size);
template void run_maxarg_reduce<float>(const float* data, data_index<float>* maxes, size_t size, size_t block_size, size_t batch_size);
template void run_maxarg_reduce<half>(const half* data, data_index<half>* maxes, size_t size, size_t block_size, size_t batch_size);

}