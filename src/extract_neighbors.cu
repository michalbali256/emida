#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "kernels.cuh"
namespace emida {

// Copies a rectangle neighborhood with size s around position determined by max_i from each subregion.
// The pixels of subregions may not be in common row-major order and pos_policy::index is used to get the actual
// index of the data with specified position in the data buffer.
template<typename T, typename pos_policy>
__global__ void extract_neighbors(const T* data, const data_index<T>* max_i, T* neighbors, int s, size2_t src_size, size_t batch_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= batch_size)
		return;

	int r = (s - 1) / 2;

	size2_t max_pos = pos_policy::shift_pos(max_i[idx].index, src_size);

	for (int i = 0; i < s; ++i)
	{
		for (int j = 0; j < s; ++j)
		{
			int from_x_i = (int)max_pos.x - r + i;
			int from_y_i = (int)max_pos.y - r + j;

			esize_t from_x = max(from_x_i, 0);
			from_x = min(from_x, src_size.x);

			esize_t from_y = max(from_y_i, 0);
			from_y = min(from_y, src_size.y);
			

			neighbors[idx * s * s + j * s + i] = data[pos_policy::index(idx, {from_x, from_y}, src_size)];
		}
	}
}

template<typename T, typename pos_policy>
void run_extract_neighbors(const T* data, const data_index<T>* max_i, T* neighbors, int s, size2_t src_size, esize_t batch_size)
{
	esize_t block_size = 128;
	esize_t grid_size = div_up(batch_size, block_size);
	extract_neighbors<T, pos_policy> <<<grid_size, block_size >>> (data, max_i, neighbors, s, src_size, batch_size);
}

template void run_extract_neighbors<double, cross_res_pos_policy_id>(const double* data, const data_index<double>* max_i, double* neighbors, int s, size2_t src_size, esize_t batch_size);
template void run_extract_neighbors<double, cross_res_pos_policy_fft>(const double* data, const data_index<double>* max_i, double* neighbors, int s, size2_t src_size, esize_t batch_size);
template void run_extract_neighbors<float, cross_res_pos_policy_id>(const float* data, const data_index<float>* max_i, float* neighbors, int s, size2_t src_size, esize_t batch_size);
template void run_extract_neighbors<float, cross_res_pos_policy_fft>(const float* data, const data_index<float>* max_i, float* neighbors, int s, size2_t src_size, esize_t batch_size);

}